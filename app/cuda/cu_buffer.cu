#include "cu_buffer.cuh"

#include "../utils.hpp"

#include <hip/hip_runtime.h>

namespace cuda {

void Buffer::allocate() {
  SPD_TRACE_FUNC;
  hipMallocManaged(&mapped_data_, size_);
}

void Buffer::free() {
  SPD_TRACE_FUNC;
  hipFree(mapped_data_);
}

} // namespace cuda
