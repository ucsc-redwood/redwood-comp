#include "hip/hip_runtime.h"
#include "device_kernels.cuh"

namespace cuda {

namespace kernels {

__global__ void vector_add(const int *input_a, const int *input_b, int *output,
                           size_t start, size_t end) {
  const auto i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i >= start && i < end) {
    output[i] = input_a[i] + input_b[i];
  }
}

} // namespace kernels

} // namespace cuda
