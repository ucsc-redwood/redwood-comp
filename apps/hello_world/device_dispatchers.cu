#include "device_dispatchers.cuh"
#include "device_kernels.cuh"

namespace cuda {

void run_stage1(const cuda::AppData &app_data) {
  // use 256 threads per block
  constexpr dim3 threads_per_block(256);
  const dim3 blocks((app_data.n + threads_per_block.x - 1) /
                    threads_per_block.x);

  cuda::kernels::vector_add<<<blocks, threads_per_block, 0>>>(
      app_data.input_a->data(), app_data.input_b->data(),
      app_data.output->data(), 0, app_data.n);

  hipDeviceSynchronize();
}

} // namespace cuda
