#include "hip/hip_runtime.h"
#include <spdlog/spdlog.h>

#include <cub/util_math.cuh>
#include <numeric>

#include "01_morton.cuh"
#include "02_sort.cuh"
#include "03_unique.cuh"
#include "04_radix_tree.cuh"
#include "05_edge_count.cuh"
#include "06_prefix_sum.cuh"
#include "07_octree.cuh"
#include "agents/prefix_sum_agent.cuh"
#include "agents/unique_agent.cuh"
#include "common.cuh"
#include "cu_dispatcher.cuh"
#include "im_storage.cuh"
#include "redwood/cuda/helpers.cuh"

namespace cuda {

namespace kernels {

// Number of digit bins
constexpr auto RADIX = 256;
constexpr auto RADIX_MASK = RADIX - 1;
constexpr auto RADIX_LOG = 8;

constexpr auto SEC_RADIX_START = 1 * RADIX;     // 256
constexpr auto THIRD_RADIX_START = 2 * RADIX;   // 512
constexpr auto FOURTH_RADIX_START = 3 * RADIX;  // 768

// For the upfront global histogram kernel
// #define G_HIST_PART_SIZE 65536
// #define G_HIST_VEC_SIZE 16384

constexpr auto G_HIST_PART_SIZE = 65536;
constexpr auto G_HIST_VEC_SIZE = 16384;

// For the digit binning
// Partition tile size in k_DigitBinning

// #define BIN_PART_SIZE 7680

// looks like we use 512 threads per block, and 15 items per thread, so 7680
constexpr auto BIN_WARPS = 512 / LANE_COUNT;  // 16;
constexpr auto BIN_KEYS_PER_THREAD = 15;
constexpr auto BIN_SUB_PART_SIZE = BIN_KEYS_PER_THREAD * LANE_COUNT;  // 480;

constexpr auto BIN_PART_SIZE = 512 * BIN_KEYS_PER_THREAD;  // 7680;

// Total size of warp histograms in shared memory in k_DigitBinning
// #define BIN_HISTS_SIZE 4096
constexpr auto BIN_HISTS_SIZE = 4096;

// Subpartition tile size of a single warp in k_DigitBinning
// #define BIN_SUB_PART_SIZE 480

// Warps per threadblock in k_DigitBinning
// Keys per thread in k_DigitBinning

// Starting offset of a subpartition tile
#define BIN_SUB_PART_START (WARP_INDEX * BIN_SUB_PART_SIZE)

// Starting offset of a partition tile
#define BIN_PART_START (partitionIndex * BIN_PART_SIZE)

// Flag value inidicating neither inclusive sum, nor reduction of a
// partition tile is ready
// for the chained scan with decoupled lookback
#define FLAG_NOT_READY 0
// Flag value indicating reduction of a partition tile is ready
#define FLAG_REDUCTION 1
// Flag value indicating inclusive sum of a partition tile is ready
#define FLAG_INCLUSIVE 2
#define FLAG_MASK 3

// ============================================================================
// Kernel entry points
// ============================================================================

__global__ void k_GlobalHistogram(const unsigned int *sort,
                                  unsigned int *global_histogram,
                                  unsigned int size) {
  __shared__ unsigned int s_globalHistFirst[RADIX * 2];
  __shared__ unsigned int s_globalHistSec[RADIX * 2];
  __shared__ unsigned int s_globalHistThird[RADIX * 2];
  __shared__ unsigned int s_globalHistFourth[RADIX * 2];

  const auto logicalBlocks = hipcub::DivideAndRoundUp(size, G_HIST_PART_SIZE);

  for (auto yanwen_block_id = blockIdx.x; yanwen_block_id < logicalBlocks;
       yanwen_block_id += gridDim.x) {
    // clear shared memory
    for (unsigned int i = threadIdx.x; i < RADIX * 2; i += blockDim.x) {
      s_globalHistFirst[i] = 0;
      s_globalHistSec[i] = 0;
      s_globalHistThird[i] = 0;
      s_globalHistFourth[i] = 0;
    }
    __syncthreads();

    // histogram
    {
      // 64 threads : 1 histogram in shared memory

      // clang-format off
       unsigned int* s_wavesHistFirst = &s_globalHistFirst[threadIdx.x / 64 * RADIX];
       unsigned int* s_wavesHistSec = &s_globalHistSec[threadIdx.x / 64 * RADIX];
       unsigned int* s_wavesHistThird = &s_globalHistThird[threadIdx.x / 64 * RADIX];
       unsigned int* s_wavesHistFourth = &s_globalHistFourth[threadIdx.x / 64 * RADIX];
      // clang-format on

      if (yanwen_block_id < logicalBlocks - 1) {
        const unsigned int partEnd = (yanwen_block_id + 1) * G_HIST_VEC_SIZE;
        for (unsigned int i = threadIdx.x + (yanwen_block_id * G_HIST_VEC_SIZE);
             i < partEnd;
             i += blockDim.x) {
          uint4 t[1] = {reinterpret_cast<const uint4 *>(sort)[i]};

          atomicAdd(&s_wavesHistFirst[reinterpret_cast<uint8_t *>(t)[0]], 1);
          atomicAdd(&s_wavesHistSec[reinterpret_cast<uint8_t *>(t)[1]], 1);
          atomicAdd(&s_wavesHistThird[reinterpret_cast<uint8_t *>(t)[2]], 1);
          atomicAdd(&s_wavesHistFourth[reinterpret_cast<uint8_t *>(t)[3]], 1);

          atomicAdd(&s_wavesHistFirst[reinterpret_cast<uint8_t *>(t)[4]], 1);
          atomicAdd(&s_wavesHistSec[reinterpret_cast<uint8_t *>(t)[5]], 1);
          atomicAdd(&s_wavesHistThird[reinterpret_cast<uint8_t *>(t)[6]], 1);
          atomicAdd(&s_wavesHistFourth[reinterpret_cast<uint8_t *>(t)[7]], 1);

          atomicAdd(&s_wavesHistFirst[reinterpret_cast<uint8_t *>(t)[8]], 1);
          atomicAdd(&s_wavesHistSec[reinterpret_cast<uint8_t *>(t)[9]], 1);
          atomicAdd(&s_globalHistThird[reinterpret_cast<uint8_t *>(t)[10]], 1);
          atomicAdd(&s_wavesHistFourth[reinterpret_cast<uint8_t *>(t)[11]], 1);

          atomicAdd(&s_wavesHistFirst[reinterpret_cast<uint8_t *>(t)[12]], 1);
          atomicAdd(&s_wavesHistSec[reinterpret_cast<uint8_t *>(t)[13]], 1);
          atomicAdd(&s_wavesHistThird[reinterpret_cast<uint8_t *>(t)[14]], 1);
          atomicAdd(&s_wavesHistFourth[reinterpret_cast<uint8_t *>(t)[15]], 1);
        }
      }

      if (yanwen_block_id == logicalBlocks - 1) {
        for (unsigned int i =
                 threadIdx.x + (yanwen_block_id * G_HIST_PART_SIZE);
             i < size;
             i += blockDim.x) {
          unsigned int t[1] = {sort[i]};
          atomicAdd(&s_wavesHistFirst[reinterpret_cast<uint8_t *>(t)[0]], 1);
          atomicAdd(&s_wavesHistSec[reinterpret_cast<uint8_t *>(t)[1]], 1);
          atomicAdd(&s_wavesHistThird[reinterpret_cast<uint8_t *>(t)[2]], 1);
          atomicAdd(&s_wavesHistFourth[reinterpret_cast<uint8_t *>(t)[3]], 1);
        }
      }
    }
    __syncthreads();

    // reduce and add to device
    for (unsigned int i = threadIdx.x; i < RADIX; i += blockDim.x) {
      atomicAdd(&global_histogram[i],
                s_globalHistFirst[i] + s_globalHistFirst[i + RADIX]);
      atomicAdd(&global_histogram[i + SEC_RADIX_START],
                s_globalHistSec[i] + s_globalHistSec[i + RADIX]);
      atomicAdd(&global_histogram[i + THIRD_RADIX_START],
                s_globalHistThird[i] + s_globalHistThird[i + RADIX]);
      atomicAdd(&global_histogram[i + FOURTH_RADIX_START],
                s_globalHistFourth[i] + s_globalHistFourth[i + RADIX]);
    }
  }
}

// fixed to use 4 blocks, and 'radix' (256) threads
__global__ void k_Scan(const unsigned int *globalHistogram,
                       unsigned int *firstPassHistogram,
                       unsigned int *secPassHistogram,
                       unsigned int *thirdPassHistogram,
                       unsigned int *fourthPassHistogram) {
  __shared__ unsigned int s_scan[RADIX];

  s_scan[threadIdx.x] = InclusiveWarpScanCircularShift(
      globalHistogram[threadIdx.x + blockIdx.x * RADIX]);
  __syncthreads();

  if (threadIdx.x < (RADIX >> LANE_LOG))
    s_scan[threadIdx.x << LANE_LOG] =
        ActiveExclusiveWarpScan(s_scan[threadIdx.x << LANE_LOG]);
  __syncthreads();

  switch (blockIdx.x) {
    case 0:
      firstPassHistogram[threadIdx.x] =
          (s_scan[threadIdx.x] +
           (getLaneId() ? __shfl_sync(0xfffffffe, s_scan[threadIdx.x - 1], 1)
                        : 0))
              << 2 |
          FLAG_INCLUSIVE;
      break;
    case 1:
      secPassHistogram[threadIdx.x] =
          (s_scan[threadIdx.x] +
           (getLaneId() ? __shfl_sync(0xfffffffe, s_scan[threadIdx.x - 1], 1)
                        : 0))
              << 2 |
          FLAG_INCLUSIVE;
      break;
    case 2:
      thirdPassHistogram[threadIdx.x] =
          (s_scan[threadIdx.x] +
           (getLaneId() ? __shfl_sync(0xfffffffe, s_scan[threadIdx.x - 1], 1)
                        : 0))
              << 2 |
          FLAG_INCLUSIVE;
      break;
    case 3:
      fourthPassHistogram[threadIdx.x] =
          (s_scan[threadIdx.x] +
           (getLaneId() ? __shfl_sync(0xfffffffe, s_scan[threadIdx.x - 1], 1)
                        : 0))
              << 2 |
          FLAG_INCLUSIVE;
      break;
    default:
      break;
  }
}

// ============================================================================
// Yanwen's version
//
// ============================================================================

__global__ void k_DigitBinningPass(unsigned int *sort,
                                   unsigned int *alt,
                                   volatile unsigned int *passHistogram,
                                   volatile unsigned int *index,
                                   unsigned int size,
                                   unsigned int radixShift) {
  __shared__ unsigned int s_warpHistograms[BIN_PART_SIZE];
  __shared__ unsigned int s_localHistogram[RADIX];

  volatile unsigned int *s_warpHist =
      &s_warpHistograms[WARP_INDEX << RADIX_LOG];

  const auto logicalBlocks = hipcub::DivideAndRoundUp(size, BIN_PART_SIZE);

  for (auto yanwen_block_id = blockIdx.x; yanwen_block_id < logicalBlocks;
       yanwen_block_id += gridDim.x) {
    // clear shared memory
    for (unsigned int i = threadIdx.x; i < BIN_HISTS_SIZE;
         i += blockDim.x)  // unnecessary work for last partion but still a win
                           // to avoid another barrier
      s_warpHistograms[i] = 0;

    // atomically assign partition tiles
    if (threadIdx.x == 0)
      s_warpHistograms[BIN_PART_SIZE - 1] =
          atomicAdd((unsigned int *)&index[radixShift >> 3], 1);
    __syncthreads();
    const unsigned int partitionIndex = s_warpHistograms[BIN_PART_SIZE - 1];

    // To handle input sizes not perfect multiples of the partition tile size
    if (partitionIndex < logicalBlocks - 1) {
      // load keys
      unsigned int keys[BIN_KEYS_PER_THREAD];
#pragma unroll
      for (unsigned int i = 0,
                        t = getLaneId() + BIN_SUB_PART_START + BIN_PART_START;
           i < BIN_KEYS_PER_THREAD;
           ++i, t += LANE_COUNT)
        keys[i] = sort[t];

      uint16_t offsets[BIN_KEYS_PER_THREAD];

// WLMS
#pragma unroll
      for (unsigned int i = 0; i < BIN_KEYS_PER_THREAD; ++i) {
        // CUB version "match any"
        /*
        unsigned warpFlags;
        #pragma unroll
        for (int k = 0; k < RADIX_LOG; ++k)
        {
            unsigned int mask;
            unsigned int current_bit = 1 << k + radixShift;
            asm("{\n"
                "    .reg .pred p;\n"
                "    and.b32 %0, %1, %2;"
                "    setp.ne.u32 p, %0, 0;\n"
                "    vote.ballot.sync.b32 %0, p, 0xffffffff;\n"
                "    @!p not.b32 %0, %0;\n"
                "}\n" : "=r"(mask) : "r"(keys[i]), "r"(current_bit));
            warpFlags = (k == 0) ? mask : warpFlags & mask;
        }
        const unsigned int bits = __popc(warpFlags & getLaneMaskLt());
        */
        unsigned warpFlags = 0xffffffff;
#pragma unroll
        for (int k = 0; k < RADIX_LOG; ++k) {
          const bool t2 = keys[i] >> k + radixShift & 1;
          warpFlags &= (t2 ? 0 : 0xffffffff) ^ __ballot_sync(0xffffffff, t2);
        }
        const unsigned int bits = __popc(warpFlags & getLaneMaskLt());

        // An alternative, but slightly slower version.
        /*
        offsets[i] = s_warpHist[keys[i] >> radixShift & RADIX_MASK] + bits;
        __syncwarp(0xffffffff);
        if (bits == 0)
            s_warpHist[keys[i] >> radixShift & RADIX_MASK] +=
            __popc(warpFlags);
        __syncwarp(0xffffffff);
        */
        unsigned int preIncrementVal;
        if (bits == 0)
          preIncrementVal = atomicAdd(
              (unsigned int *)&s_warpHist[keys[i] >> radixShift & RADIX_MASK],
              __popc(warpFlags));

        offsets[i] =
            __shfl_sync(0xffffffff, preIncrementVal, __ffs(warpFlags) - 1) +
            bits;
      }
      __syncthreads();

      // exclusive prefix sum up the warp histograms
      if (threadIdx.x < RADIX) {
        unsigned int reduction = s_warpHistograms[threadIdx.x];
        for (unsigned int i = threadIdx.x + RADIX; i < BIN_HISTS_SIZE;
             i += RADIX) {
          reduction += s_warpHistograms[i];
          s_warpHistograms[i] = reduction - s_warpHistograms[i];
        }

        atomicAdd((unsigned int *)&passHistogram[threadIdx.x +
                                                 (partitionIndex + 1) * RADIX],
                  FLAG_REDUCTION | reduction << 2);

        // begin the exclusive prefix sum across the reductions
        s_localHistogram[threadIdx.x] =
            InclusiveWarpScanCircularShift(reduction);
      }
      __syncthreads();

      if (threadIdx.x < (RADIX >> LANE_LOG))
        s_localHistogram[threadIdx.x << LANE_LOG] =
            ActiveExclusiveWarpScan(s_localHistogram[threadIdx.x << LANE_LOG]);
      __syncthreads();

      if (threadIdx.x < RADIX && getLaneId())
        s_localHistogram[threadIdx.x] +=
            __shfl_sync(0xfffffffe, s_localHistogram[threadIdx.x - 1], 1);
      __syncthreads();

      // update offsets
      if (WARP_INDEX) {
#pragma unroll
        for (unsigned int i = 0; i < BIN_KEYS_PER_THREAD; ++i) {
          const unsigned int t2 = keys[i] >> radixShift & RADIX_MASK;
          offsets[i] += s_warpHist[t2] + s_localHistogram[t2];
        }
      } else {
#pragma unroll
        for (unsigned int i = 0; i < BIN_KEYS_PER_THREAD; ++i)
          offsets[i] += s_localHistogram[keys[i] >> radixShift & RADIX_MASK];
      }
      __syncthreads();

// scatter keys into shared memory
#pragma unroll
      for (unsigned int i = 0; i < BIN_KEYS_PER_THREAD; ++i)
        s_warpHistograms[offsets[i]] = keys[i];

      // split the warps into single thread cooperative groups and lookback
      if (threadIdx.x < RADIX) {
        unsigned int reduction = 0;
        for (unsigned int k = partitionIndex; k >= 0;) {
          const unsigned int flagPayload =
              passHistogram[threadIdx.x + k * RADIX];

          if ((flagPayload & FLAG_MASK) == FLAG_INCLUSIVE) {
            reduction += flagPayload >> 2;
            atomicAdd(
                (unsigned int *)&passHistogram[threadIdx.x +
                                               (partitionIndex + 1) * RADIX],
                1 | (reduction << 2));
            s_localHistogram[threadIdx.x] =
                reduction - s_localHistogram[threadIdx.x];
            break;
          }

          if ((flagPayload & FLAG_MASK) == FLAG_REDUCTION) {
            reduction += flagPayload >> 2;
            k--;
          }
        }
      }
      __syncthreads();

// scatter runs of keys into device memory
#pragma unroll
      for (unsigned int i = threadIdx.x; i < BIN_PART_SIZE; i += blockDim.x)
        alt[s_localHistogram[s_warpHistograms[i] >> radixShift & RADIX_MASK] +
            i] = s_warpHistograms[i];
    }

    // Process the final partition slightly differently
    if (partitionIndex == logicalBlocks - 1) {
      // immediately begin lookback
      if (threadIdx.x < RADIX) {
        if (partitionIndex) {
          unsigned int reduction = 0;
          for (unsigned int k = partitionIndex; k >= 0;) {
            const unsigned int flagPayload =
                passHistogram[threadIdx.x + k * RADIX];

            if ((flagPayload & FLAG_MASK) == FLAG_INCLUSIVE) {
              reduction += flagPayload >> 2;
              s_localHistogram[threadIdx.x] = reduction;
              break;
            }

            if ((flagPayload & FLAG_MASK) == FLAG_REDUCTION) {
              reduction += flagPayload >> 2;
              k--;
            }
          }
        } else {
          s_localHistogram[threadIdx.x] = passHistogram[threadIdx.x] >> 2;
        }
      }
      __syncthreads();

      const unsigned int partEnd = BIN_PART_START + BIN_PART_SIZE;
      for (unsigned int i = threadIdx.x + BIN_PART_START; i < partEnd;
           i += blockDim.x) {
        unsigned int key;
        unsigned int offset;
        unsigned warpFlags = 0xffffffff;

        if (i < size) key = sort[i];

#pragma unroll
        for (unsigned int k = 0; k < RADIX_LOG; ++k) {
          const bool t = key >> k + radixShift & 1;
          warpFlags &= (t ? 0 : 0xffffffff) ^ __ballot_sync(0xffffffff, t);
        }
        const unsigned int bits = __popc(warpFlags & getLaneMaskLt());

#pragma unroll
        for (unsigned int k = 0; k < BIN_WARPS; ++k) {
          unsigned int preIncrementVal;
          if (WARP_INDEX == k && bits == 0 && i < size)
            preIncrementVal =
                atomicAdd(&s_localHistogram[key >> radixShift & RADIX_MASK],
                          __popc(warpFlags));

          if (WARP_INDEX == k)
            offset =
                __shfl_sync(0xffffffff, preIncrementVal, __ffs(warpFlags) - 1) +
                bits;
          __syncthreads();
        }

        if (i < size) alt[offset] = key;
      }
    }
  }
}

// //
// ----------------------------------------------------------------------------
// // Unique
// //
// ----------------------------------------------------------------------------

// __global__ void k_FindDups(const unsigned int *u_keys,
//                            int *u_flag_heads,
//                            const int n) {
//   __shared__ agents::UniqueAgent::TempStorage temp_storage;

//   agents::UniqueAgent agent(n);
//   agent.Process_FindDups(temp_storage, u_keys, u_flag_heads, n);
// }

// __global__ void k_MoveDups(const unsigned int *u_keys,
//                            const int *u_flag_heads_sums,
//                            const int n,
//                            unsigned int *u_keys_out,
//                            int *n_unique_out) {
//   agents::UniqueAgent agent(n);
//   agent.Process_MoveDups(
//       u_keys, u_flag_heads_sums, n, u_keys_out, n_unique_out);
// }

}  // namespace kernels

// ----------------------------------------------------------------------------
// Stage 1 (input -> morton code)
// ----------------------------------------------------------------------------

void run_stage1(AppData &app_data, hipStream_t stream) {
  static constexpr auto block_size = 256;
  const auto grid_size = div_up(app_data.get_n_input(), block_size);
  constexpr auto s_mem = 0;

  spdlog::debug(
      "CUDA kernel 'compute_morton_code', n = {}, threads = {}, blocks = {}, "
      "stream: {}",
      app_data.get_n_input(),
      block_size,
      grid_size,
      reinterpret_cast<void *>(stream));

  kernels::k_ComputeMortonCode<<<grid_size, block_size, s_mem, stream>>>(
      app_data.u_input_points.data(),
      app_data.u_morton_keys.data(),
      app_data.get_n_input(),
      app_data.min_coord,
      app_data.range);

  CUDA_CHECK(hipStreamSynchronize(stream));
}

// ----------------------------------------------------------------------------
// Stage 2 (sort) (morton code -> sorted morton code)
// ----------------------------------------------------------------------------

void run_stage2(AppData &app_data, ImStorage &im_storage, hipStream_t stream) {
  const auto n = app_data.get_n_input();

  im_storage.clearSmem();

  const auto smem = 0;
  constexpr auto grid_size = 16;

  kernels::k_GlobalHistogram<<<grid_size,
                               ImStorage::GLOBAL_HIST_THREADS,
                               smem,
                               stream>>>(app_data.u_morton_keys.data(),
                                         im_storage.d_global_histogram,
                                         (int)app_data.get_n_input());

  kernels::k_Scan<<<ImStorage::RADIX_PASSES, ImStorage::RADIX, 0, stream>>>(
      im_storage.d_global_histogram,
      im_storage.d_first_pass_histogram,
      im_storage.d_second_pass_histogram,
      im_storage.d_third_pass_histogram,
      im_storage.d_fourth_pass_histogram);

  kernels::k_DigitBinningPass<<<grid_size,
                                ImStorage::BINNING_THREADS,
                                0,
                                stream>>>(
      app_data.u_morton_keys.data(),  // <---
      app_data.u_morton_keys_alt.data(),
      im_storage.d_first_pass_histogram,
      im_storage.d_index,
      n,
      0);

  kernels::
      k_DigitBinningPass<<<grid_size, ImStorage::BINNING_THREADS, 0, stream>>>(
          app_data.u_morton_keys_alt.data(),
          app_data.u_morton_keys.data(),  // <---
          im_storage.d_second_pass_histogram,
          im_storage.d_index,
          n,
          8);

  kernels::k_DigitBinningPass<<<grid_size,
                                ImStorage::BINNING_THREADS,
                                0,
                                stream>>>(
      app_data.u_morton_keys.data(),  // <---
      app_data.u_morton_keys_alt.data(),
      im_storage.d_third_pass_histogram,
      im_storage.d_index,
      n,
      16);

  kernels::
      k_DigitBinningPass<<<grid_size, ImStorage::BINNING_THREADS, 0, stream>>>(
          app_data.u_morton_keys_alt.data(),
          app_data.u_morton_keys.data(),  // <---
          im_storage.d_fourth_pass_histogram,
          im_storage.d_index,
          n,
          24);

  CUDA_CHECK(hipStreamSynchronize(stream));
}

// ----------------------------------------------------------------------------
// Stage 3 (unique) (sorted morton code -> unique sorted morton code)
// ----------------------------------------------------------------------------

void run_stage3(AppData &app_data, ImStorage &im_storage, hipStream_t stream) {
  constexpr auto unique_block_size = agents::UniqueAgent::n_threads;  // 256
  constexpr auto prefix_block_size =
      agents::PrefixSumAgent<unsigned int>::n_threads;  // 128

  constexpr auto grid_size = 16;

  kernels::k_FindDups<<<grid_size, unique_block_size, 0, stream>>>(
      app_data.get_sorted_morton_keys(),
      im_storage.u_flag_heads,  // <-- output
      app_data.get_n_input());

  CUDA_CHECK(hipStreamSynchronize(stream));

  ::std::partial_sum(im_storage.u_flag_heads,
                     im_storage.u_flag_heads + app_data.get_n_input(),
                     im_storage.u_flag_heads);

  kernels::k_MoveDups<<<grid_size, unique_block_size, 0, stream>>>(
      app_data.get_sorted_morton_keys(),
      im_storage.u_flag_heads,
      app_data.get_n_input(),
      app_data.get_unique_morton_keys(),  // <-- output
      nullptr);

  CUDA_CHECK(hipStreamSynchronize(stream));

  const auto n_unique = im_storage.u_flag_heads[app_data.get_n_input() - 1] + 1;
  app_data.set_n_unique(n_unique);
  app_data.set_n_brt_nodes(n_unique - 1);
}

// ----------------------------------------------------------------------------
// Stage 4 (build tree) (unique sorted morton code -> tree nodes)
// ----------------------------------------------------------------------------

void run_stage4(AppData &app_data, hipStream_t stream) {
  constexpr auto gridDim = 16;
  constexpr auto blockDim = 512;
  constexpr auto sharedMem = 0;

  kernels::k_BuildRadixTree<<<gridDim, blockDim, sharedMem, stream>>>(
      app_data.get_n_unique(),
      app_data.get_unique_morton_keys(),
      app_data.brt.u_prefix_n.data(),
      app_data.brt.u_has_leaf_left.data(),
      app_data.brt.u_has_leaf_right.data(),
      app_data.brt.u_left_child.data(),
      app_data.brt.u_parents.data());

  CUDA_CHECK(hipStreamSynchronize(stream));
}

// ----------------------------------------------------------------------------
// Stage 5 (edge count) (tree nodes -> edge count)
// ----------------------------------------------------------------------------

void run_stage5(AppData &app_data, hipStream_t stream) {
  constexpr auto gridDim = 16;
  constexpr auto blockDim = 512;
  constexpr auto sharedMem = 0;

  kernels::k_EdgeCount<<<gridDim, blockDim, sharedMem, stream>>>(
      app_data.brt.u_prefix_n.data(),
      app_data.brt.u_parents.data(),
      app_data.u_edge_count.data(),
      app_data.get_n_brt_nodes());

  CUDA_CHECK(hipStreamSynchronize(stream));
}

// ----------------------------------------------------------------------------
// Stage 6 (edge offset) (edge count -> edge offset)
// ----------------------------------------------------------------------------

void run_stage6(AppData &app_data, hipStream_t stream) {
  constexpr auto n_threads = agents::PrefixSumAgent<int>::n_threads;

  // has to be single block
  kernels::k_SingleBlockExclusiveScan<<<1, n_threads, 0, stream>>>(
      app_data.u_edge_count.data(),
      app_data.u_edge_offset.data(),  // <-- output
      app_data.get_n_brt_nodes());

  CUDA_CHECK(hipStreamSynchronize(stream));

  // num oct is the result of last of prefix sum
  app_data.set_n_octree_nodes(
      app_data.u_edge_offset[app_data.get_n_brt_nodes() - 1]);
}

// ----------------------------------------------------------------------------
// Stage 7 (octree) (everything above -> octree)
// ----------------------------------------------------------------------------

void run_stage7(AppData &app_data, hipStream_t stream) {
  constexpr auto block_size = 512;
  //  k_MakeOctNodes<<<grid_size, block_size, 0, stream>>>(
  //       pipe->oct.u_children,
  //       pipe->oct.u_corner,
  //       pipe->oct.u_cell_size,
  //       pipe->oct.u_child_node_mask,
  //       pipe->u_edge_offsets,
  //       pipe->u_edge_counts,
  //       pipe->getUniqueKeys(),
  //       pipe->brt.u_prefix_n,
  //       pipe->brt.u_parents,
  //       pipe->min_coord,
  //       pipe->range,
  //       pipe->n_brt_nodes());

  //   k_LinkLeafNodes<<<grid_size, block_size, 0, stream>>>(
  //       pipe->oct.u_children,
  //       pipe->oct.u_child_node_mask,
  //       pipe->u_edge_offsets,
  //       pipe->u_edge_counts,
  //       pipe->getUniqueKeys(),
  //       pipe->brt.u_has_leaf_left,
  //       pipe->brt.u_has_leaf_right,
  //       pipe->brt.u_prefix_n,
  //       pipe->brt.u_parents,
  //       pipe->brt.u_left_child,
  //       pipe->n_brt_nodes());
  constexpr auto gridDim = 16;
  constexpr auto blockDim = 512;
  constexpr auto sharedMem = 0;

  kernels::k_MakeOctNodes<<<gridDim, blockDim, sharedMem, stream>>>(
      reinterpret_cast<int(*)[8]>(app_data.oct.u_children.data()),
      app_data.oct.u_corner.data(),
      app_data.oct.u_cell_size.data(),
      app_data.oct.u_child_node_mask.data(),
      app_data.u_edge_offset.data(),
      app_data.u_edge_count.data(),
      app_data.get_unique_morton_keys(),
      app_data.brt.u_prefix_n.data(),
      app_data.brt.u_parents.data(),
      app_data.min_coord,
      app_data.range,
      app_data.get_n_brt_nodes());

  //   kernels::k_LinkLeafNodes<<<gridDim, blockDim, sharedMem, stream>>>(
  //       reinterpret_cast<int(*)[8]>(app_data.oct.u_children.data()),
  //       app_data.oct.u_child_node_mask.data(),
  //       app_data.u_edge_offset.data(),
  //       app_data.u_edge_count.data(),
  //       app_data.get_unique_morton_keys(),
  //       app_data.brt.u_has_leaf_left.data(),
  //       app_data.brt.u_has_leaf_right.data(),
  //       app_data.brt.u_prefix_n.data(),
  //       app_data.brt.u_parents.data(),
  //       app_data.brt.u_left_child.data(),
  //       app_data.get_n_brt_nodes());

  CUDA_CHECK(hipStreamSynchronize(stream));
}

}  // namespace cuda
