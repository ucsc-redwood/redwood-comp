#include <spdlog/spdlog.h>

#include "cu_dispatcher.cuh"
#include "cu_kernels.cuh"
#include "redwood/cuda/helpers.cuh"

namespace cuda {

// #define LOG_KERNEL(NAME)                                                 \
//   spdlog::debug(                                                         \
//       "CUDA kernel '{}', n = {}, threads = {}, blocks = {}, stream: {}", \
//       NAME,                                                              \
//       total_iterations,                                                  \
//       block_dim,                                                         \
//       grid_dim,                                                          \
//       reinterpret_cast<void *>(stream));

// -----------------------------------------------------------------------------
// Stage 1 (first conv2d)
// -----------------------------------------------------------------------------

void run_stage1(AppData &app_data, const hipStream_t stream, bool sync) {
  static const auto total_iterations =
      model::kConv1OutChannels * model::kConv1OutHeight * model::kConv1OutWidth;

  static constexpr auto block_dim = dim3{256, 1, 1};
  static const auto grid_dim = div_up(total_iterations, block_dim.x);
  static constexpr auto shared_mem = 0;

  kernels::dense::conv2d<<<grid_dim, block_dim, shared_mem, stream>>>(
      app_data.u_image.data(),
      app_data.u_conv1_weights.data(),
      app_data.u_conv1_bias.data(),
      app_data.u_conv1_out.data(),
      model::kInputHeight,
      model::kInputWidth,
      model::kConv1OutChannels,
      model::kInputChannels,
      model::kKernelSize,
      model::kKernelSize,
      model::kConv1BiasSize,
      model::kKernelSize,
      model::kStride,
      model::kPadding,
      model::kConv1OutHeight,
      model::kConv1OutWidth,
      model::kRelu);

  if (sync) {
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

// -----------------------------------------------------------------------------
// Stage 2 (maxpool)
// -----------------------------------------------------------------------------

void run_stage2(AppData &app_data, const hipStream_t stream, bool sync) {
  static const auto total_iterations =
      model::kConv1OutChannels * model::kPool1OutHeight * model::kPool1OutWidth;

  static constexpr auto block_dim = dim3{256, 1, 1};
  static const auto grid_dim = div_up(total_iterations, block_dim.x);
  static constexpr auto shared_mem = 0;

  //   LOG_KERNEL("maxpool2d");

  kernels::dense::maxpool2d<<<grid_dim, block_dim, shared_mem, stream>>>(
      app_data.u_conv1_out.data(),
      app_data.u_pool1_out.data(),
      model::kConv1OutChannels,
      model::kConv1OutHeight,
      model::kConv1OutWidth,
      model::kPoolSize,
      model::kStride,
      model::kPool1OutHeight,
      model::kPool1OutWidth);

  if (sync) {
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

// -----------------------------------------------------------------------------
// Stage 3 (second conv2d)
// -----------------------------------------------------------------------------

void run_stage3(AppData &app_data, const hipStream_t stream, bool sync) {
  static const auto total_iterations =
      model::kConv2OutChannels * model::kConv2OutHeight * model::kConv2OutWidth;

  static constexpr auto block_dim = dim3{256, 1, 1};
  static const auto grid_dim = div_up(total_iterations, block_dim.x);
  static constexpr auto shared_mem = 0;

  //   LOG_KERNEL("conv2d");

  kernels::dense::conv2d<<<grid_dim, block_dim, shared_mem, stream>>>(
      app_data.u_pool1_out.data(),
      app_data.u_conv2_weights.data(),
      app_data.u_conv2_bias.data(),
      app_data.u_conv2_out.data(),
      model::kPool1OutHeight,
      model::kPool1OutWidth,
      model::kConv2OutChannels,
      model::kConv1OutChannels,
      model::kKernelSize,
      model::kKernelSize,
      model::kConv2BiasSize,
      model::kKernelSize,
      model::kStride,
      model::kPadding,
      model::kConv2OutHeight,
      model::kConv2OutWidth,
      model::kRelu);

  if (sync) {
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

// -----------------------------------------------------------------------------
// Stage 4 (second maxpool2d)
// -----------------------------------------------------------------------------

void run_stage4(AppData &app_data, const hipStream_t stream, bool sync) {
  static const auto total_iterations =
      model::kConv2OutChannels * model::kPool2OutHeight * model::kPool2OutWidth;

  static constexpr auto block_dim = dim3{256, 1, 1};
  static const auto grid_dim = div_up(total_iterations, block_dim.x);
  static constexpr auto shared_mem = 0;

  //   LOG_KERNEL("maxpool2d");

  kernels::dense::maxpool2d<<<grid_dim, block_dim, shared_mem, stream>>>(
      app_data.u_conv2_out.data(),
      app_data.u_pool2_out.data(),
      model::kConv2OutChannels,
      model::kConv2OutHeight,
      model::kConv2OutWidth,
      model::kPoolSize,
      model::kStride,
      model::kPool2OutHeight,
      model::kPool2OutWidth);

  if (sync) {
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

// -----------------------------------------------------------------------------
// Stage 5 (third conv2d)
// -----------------------------------------------------------------------------

void run_stage5(AppData &app_data, const hipStream_t stream, bool sync) {
  static const auto total_iterations =
      model::kConv3OutChannels * model::kConv3OutHeight * model::kConv3OutWidth;

  static constexpr auto block_dim = dim3{256, 1, 1};
  static const auto grid_dim = div_up(total_iterations, block_dim.x);
  static constexpr auto shared_mem = 0;

  //   LOG_KERNEL("conv2d");

  kernels::dense::conv2d<<<grid_dim, block_dim, shared_mem, stream>>>(
      app_data.u_pool2_out.data(),
      app_data.u_conv3_weights.data(),
      app_data.u_conv3_bias.data(),
      app_data.u_conv3_out.data(),
      model::kPool2OutHeight,
      model::kPool2OutWidth,
      model::kConv3OutChannels,
      model::kConv2OutChannels,
      model::kKernelSize,
      model::kKernelSize,
      model::kConv3BiasSize,
      model::kKernelSize,
      model::kStride,
      model::kPadding,
      model::kConv3OutHeight,
      model::kConv3OutWidth,
      model::kRelu);

  if (sync) {
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

// -----------------------------------------------------------------------------
// Stage 6 (fourth conv2d)
// -----------------------------------------------------------------------------

void run_stage6(AppData &app_data, const hipStream_t stream, bool sync) {
  static const auto total_iterations =
      model::kConv4OutChannels * model::kConv4OutHeight * model::kConv4OutWidth;

  static constexpr auto block_dim = dim3{256, 1, 1};
  static const auto grid_dim = div_up(total_iterations, block_dim.x);
  static constexpr auto shared_mem = 0;

  //   LOG_KERNEL("conv2d");

  kernels::dense::conv2d<<<grid_dim, block_dim, shared_mem, stream>>>(
      app_data.u_conv3_out.data(),
      app_data.u_conv4_weights.data(),
      app_data.u_conv4_bias.data(),
      app_data.u_conv4_out.data(),
      model::kConv3OutHeight,
      model::kConv3OutWidth,
      model::kConv4OutChannels,
      model::kConv3OutChannels,
      model::kKernelSize,
      model::kKernelSize,
      model::kConv4BiasSize,
      model::kKernelSize,
      model::kStride,
      model::kPadding,
      model::kConv4OutHeight,
      model::kConv4OutWidth,
      model::kRelu);

  if (sync) {
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

// -----------------------------------------------------------------------------
// Stage 7 (fifth conv2d)
// -----------------------------------------------------------------------------

void run_stage7(AppData &app_data, const hipStream_t stream, bool sync) {
  static const auto total_iterations =
      model::kConv5OutChannels * model::kConv5OutHeight * model::kConv5OutWidth;

  static constexpr auto block_dim = dim3{256, 1, 1};
  static const auto grid_dim = div_up(total_iterations, block_dim.x);
  static constexpr auto shared_mem = 0;

  //   LOG_KERNEL("conv2d");

  kernels::dense::conv2d<<<grid_dim, block_dim, shared_mem, stream>>>(
      app_data.u_conv4_out.data(),
      app_data.u_conv5_weights.data(),
      app_data.u_conv5_bias.data(),
      app_data.u_conv5_out.data(),
      model::kConv4OutHeight,
      model::kConv4OutWidth,
      model::kConv5OutChannels,
      model::kConv4OutChannels,
      model::kKernelSize,
      model::kKernelSize,
      model::kConv5BiasSize,
      model::kKernelSize,
      model::kStride,
      model::kPadding,
      model::kConv5OutHeight,
      model::kConv5OutWidth,
      model::kRelu);

  if (sync) {
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

// -----------------------------------------------------------------------------
// Stage 8 (third maxpool2d)
// -----------------------------------------------------------------------------

void run_stage8(AppData &app_data, const hipStream_t stream, bool sync) {
  static const auto total_iterations =
      model::kConv5OutChannels * model::kPool3OutHeight * model::kPool3OutWidth;

  static constexpr auto block_dim = dim3{256, 1, 1};
  static const auto grid_dim = div_up(total_iterations, block_dim.x);
  static constexpr auto shared_mem = 0;

  //   LOG_KERNEL("maxpool2d");

  kernels::dense::maxpool2d<<<grid_dim, block_dim, shared_mem, stream>>>(
      app_data.u_conv5_out.data(),
      app_data.u_pool3_out.data(),
      model::kConv5OutChannels,
      model::kConv5OutHeight,
      model::kConv5OutWidth,
      model::kPoolSize,
      model::kStride,
      model::kPool3OutHeight,
      model::kPool3OutWidth);

  if (sync) {
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

// -----------------------------------------------------------------------------
// Stage 9 (linear)
// -----------------------------------------------------------------------------

void run_stage9(AppData &app_data, const hipStream_t stream, bool sync) {
  static const auto total_iterations = model::kLinearOutFeatures;

  static constexpr auto block_dim = dim3{256, 1, 1};
  static const auto grid_dim = div_up(total_iterations, block_dim.x);
  static constexpr auto shared_mem = 0;

  //   LOG_KERNEL("linear");

  kernels::dense::linear<<<grid_dim, block_dim, shared_mem, stream>>>(
      app_data.u_pool3_out.data(),
      app_data.u_linear_weights.data(),
      app_data.u_linear_bias.data(),
      app_data.u_linear_out.data(),
      model::kLinearInFeatures,
      model::kLinearOutFeatures);

  if (sync) {
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

}  // namespace cuda
