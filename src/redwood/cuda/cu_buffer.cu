#include "cu_buffer.cuh"
#include "cu_helpers.cuh"

CudaBuffer::CudaBuffer(uint32_t size) : UsmBuffer(size) {
  CUDA_CHECK(hipMallocManaged(&u_data, size));
}

CudaBuffer::~CudaBuffer() { CUDA_CHECK(hipFree(u_data)); }
