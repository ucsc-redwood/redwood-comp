#include <hip/hip_runtime.h>
#include <spdlog/spdlog.h>

#include "cu_mem_resource.cuh"
#include "helpers.cuh"

namespace cuda {

void* CudaMemoryResource::do_allocate(std::size_t bytes, std::size_t) {
  SPDLOG_TRACE(
      "{}(): Allocating {} bytes at address {}", __func__, bytes, (void*)ptr);

  void* ptr = nullptr;
  CUDA_CHECK(hipMallocManaged(&ptr, bytes));
  return ptr;
}

void CudaMemoryResource::do_deallocate(void* p, std::size_t, std::size_t) {
  SPDLOG_TRACE("{}(): Deallocating address {}", __func__, (void*)p);

  CUDA_CHECK(hipFree(p));
}

bool CudaMemoryResource::do_is_equal(
    const memory_resource& other) const noexcept {
  return dynamic_cast<const CudaMemoryResource*>(&other) != nullptr;
}

}  // namespace cuda
